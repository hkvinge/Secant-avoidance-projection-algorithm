#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <cmath>
#include "kernels.cuh"
using namespace std;
		    
__global__ void readVec(float * dvector){
	/**
	Read some number (specified by the number of threads at execution)
	of entries from an array of floats on the device. Useful to debugging.

	@param dvector The name of the array of floats on the device that you
	wish to read entries from
	*/
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	printf("The entry at %d is %f\n",idx,dvector[idx]);
}

__global__ void calculate_secants(float * dsecants_out, float * dpoints_in, int * dsize_constants_in){
	/** 
	Read in a set of data points given as a 1-dimensional float array and output 
	1-dimensional float array with all normalized secants between these points.
	Assume that there are N points which live in dimension D. Then there will
	be S := N(N-1)/2 secants. In order to bijectively assign a thread id 
	(we assume that the kernel is called so that the number of threads matches
	the number of secants S), we use the map which takes 
	an integer k from {1,2,...,S} and assigns it to (i,j) such that
	k = Nj + i. If 

	@param dpoints_in The array of floats giving the collection of data points
	in question. Note that we stack all points so that the length of this
	array should be (number of points)*(dimenion of space where points live).

	@param dsecants_out The array of floats giving the collection of normalized
	secants between all data points in dpoints_in. Note that we stack all 
	secants so that if we originally have N points, the length of dsecants_out is 
	N(N-1)/2 * (dimension of space where points live).

	@param dsize_constants_in An array of integers holding important size 
	constants. 
		dsize_constants_in[0] = dimension where points live
		dsize_constants_in[1] = number of points.

	*/
	
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	// The number of data points
	int n = dsize_constants_in[1];
	// Dimension where data lives
	int data_dim = dsize_constants_in[0];
	// Coordinates for assigning a thread to a pair of data points
	int i = idx % (n-1);
	int j = (idx - i)/(n-1);
	// Index for first data point in pair
	int pair1;
	// Index for second data point in pair
	int pair2;

	if (i >= j){
		pair1 = i+1;
		pair2 = j;
	}else{
		pair1 = n-i-1;
		pair2 = n-j-1;
	}
	for (int p = 0; p < data_dim; p++){
		dsecants_out[idx*data_dim + p] = dpoints_in[data_dim*pair1 + p] - dpoints_in[data_dim*pair2 + p];
	}
	float norm = 0;
	// Calculate the norm of the secant in order to normalize it
	for (int p = 0; p < data_dim; p++){
		norm = norm + powf(dsecants_out[idx*data_dim + p],2);
	}
	norm = sqrtf(norm);
	// If the length is greater than zero, normalize the secant
	// and assign it to the secant array.
	if (norm != 0.0){
		for (int p = 0; p < data_dim; p++){
			dsecants_out[idx*data_dim + p] = (1/norm)*dsecants_out[idx*data_dim + p];
		}
	}
}

// Take a matrix and return a vector whose entries are the l2 norms of the matrix
__global__ void calculate_col_norms(float * dprojected_secants, float * dsecant_norms, int * dsize_constants_in){
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	int proj_dim = dsize_constants_in[2];
	float sum = 0;
	for (int i = 0; i < proj_dim; i++){
		sum = sum + powf(dprojected_secants[idx*proj_dim + i],2);
	}
	dsecant_norms[idx] = sqrtf(sum);
}

// Switch two columns in the matrix
__global__ void switch_columns(float * matrix, int * dcolumn_switch_constants){
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	float temp;
	temp = matrix[(dcolumn_switch_constants[1])*dcolumn_switch_constants[0] + idx];
	matrix[(dcolumn_switch_constants[1])*dcolumn_switch_constants[0] + idx] =  matrix[(dcolumn_switch_constants[2])*dcolumn_switch_constants[0] + idx];
	matrix[(dcolumn_switch_constants[2])*dcolumn_switch_constants[0] + idx] = temp;
}

// Shift vector in first column of projection
__global__ void shift_first_column(float * matrix, float * projection,float * shortest_secant, float * dalgo_constant){
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	float alpha = dalgo_constant[0];
	matrix[idx] = (1.0-alpha)*projection[idx] + alpha*(shortest_secant[idx] - projection[idx]);
}

// Normalize first column
__global__ void normalize_first_column(float * dproj, int * dsize_constants_in){
	float sum = 0;
	int column_height = dsize_constants_in[0];
	for (int i = 0; i < column_height; i++){
		sum = sum + powf(dproj[i],2);
	}
	sum = sqrtf(sum);
	if (sum != 0.0){
		for (int i = 0; i < column_height; i++){
			dproj[i] = (1/sum)*dproj[i];
		}
	}	
}

// Take a given matrix and turn it into an identity matrix
__global__ void make_identity(float * didentity_matrix, int * dsize_constants_in){
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	int col = idx/dsize_constants_in[2];
	int row = idx - col*dsize_constants_in[2];
	if (row == col){
		didentity_matrix[idx] = 1.0;
	}else{
		didentity_matrix[idx] = 0.0;
	}
}
