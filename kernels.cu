#include "hip/hip_runtime.h"
/*! \file
*  \brief Custom GPU kernels used in the SAP algorithm
*
*/

#include <stdio.h>
#include <stdlib.h>
#include <cmath>
#include "kernels.cuh"
using namespace std;


__global__ void readVec(float * d_vector){
	/**
	Reads some number (specified by the number of threads at execution)
	of entries from an array of floats on the device. Useful for debugging.

	@param d_vector The name of the array of floats on the device
	which the kernel prints entries from. 
	*/
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	printf("The entry at %d is %f\n",idx,dvector_in[idx]);
}

__global__ void readVecInt(int * d_vector){
	/**
	Reads some number (specified by the number of threads at execution)
	of entries from an array of floats on the device. Useful for debugging.

	@param d_vector The name of the array of floats on the device
	which the kernel prints entries from. 
	*/
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	printf("The entry at %d is %i\n",idx,d_vector[idx]);
}

// Calculate the secants for a collection of vectors stored as columns in a matrix, then normalize
__global__ void calculate_secants(float * d_secants, float * dpoints_in, int * dsize_constants_in){
	/** 
	Calculates the normalized secant set for a set of points.

	@param d_secants The secant set for d_points. This is the output.
	@param d_points The input points.
	@param d_int_constants An integer array which holds the input dimension 
	and number of points

	*/

	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	// Number of points
	int n = dsize_constants_in[1];
	// Input dimension
	int input_dim = dsize_constants_in[0];
	// Parameters used to pair points to calculate secants
	int i = idx % (n-1);
	int j = (idx - i)/(n-1);
	// Number of points for given pair
	int pair1;
	int pair2;
	if (i >= j){
		pair1 = i+1;
		pair2 = j;
	}else{
		pair1 = n-i-1;
		pair2 = n-j-1;
	}
	// For loop calculates secant coordinate by coordinate
	for (int p = 0; p < input_dim; p++){
		d_secants[idx*input_dim + p] = dpoints_in[input_dim*pair1 + p] - dpoints_in[input_dim*pair2 + p];
	}
	// Variable to store the norm of the secant
	float norm = 0;
	// Iterate through entries of the secant to calculate its norm
	for (int p = 0; p < input_dim; p++){
		norm = norm + powf(d_secants[idx*input_dim + p],2);
	}
	norm = sqrtf(norm);
	// As long as the norm is not zero, normalize the secant
	if (norm != 0.0){
		for (int p = 0; p < input_dim; p++){
			d_secants[idx*input_dim + p] = (1/norm)*d_secants[idx*input_dim + p];
		}
	}
}

// Take a matrix and return a vector whose entries are the l2 norms of the matrix
__global__ void calculate_col_norms(float * dprojected_secants, float * dsecant_norms, int * dsize_constants_in){
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	int proj_dim = dsize_constants_in[2];
	float sum = 0;
	for (int i = 0; i < proj_dim; i++){
		sum = sum + powf(dprojected_secants[idx*proj_dim + i],2);
	}
	dsecant_norms[idx] = sqrtf(sum);
}

// Switch two columns in the matrix
__global__ void switch_columns(float * matrix, int * dcolumn_switch_constants){
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	float temp;
	temp = matrix[(dcolumn_switch_constants[1])*dcolumn_switch_constants[0] + idx];
	matrix[(dcolumn_switch_constants[1])*dcolumn_switch_constants[0] + idx] =  matrix[(dcolumn_switch_constants[2])*dcolumn_switch_constants[0] + idx];
	matrix[(dcolumn_switch_constants[2])*dcolumn_switch_constants[0] + idx] = temp;
}

// Shift vector in first column of projection
__global__ void shift_first_column(float * matrix, float * projection,float * shortest_secant, float * dalgo_constant){
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	float alpha = dalgo_constant[0];
	matrix[idx] = (1.0-alpha)*projection[idx] + alpha*(shortest_secant[idx] - projection[idx]);
}

// Normalize first column
__global__ void normalize_first_column(float * dproj, int * dsize_constants_in){
	float sum = 0;
	int column_height = dsize_constants_in[0];
	for (int i = 0; i < column_height; i++){
		sum = sum + powf(dproj[i],2);
	}
	sum = sqrtf(sum);
	if (sum != 0.0){
		for (int i = 0; i < column_height; i++){
			dproj[i] = (1/sum)*dproj[i];
		}
	}	
}

// Take a given matrix and turn it into an identity matrix
__global__ void make_identity(float * didentity_matrix, int * dsize_constants_in){
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	int col = idx/dsize_constants_in[2];
	int row = idx - col*dsize_constants_in[2];
	if (row == col){
		didentity_matrix[idx] = 1.0;
	}else{
		didentity_matrix[idx] = 0.0;
	}
}









